#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h> // Include cuRAND header
#include <stdio.h>

// Kernel to initialize cuRAND states
__global__ void setup_kernel(hiprandState* state, unsigned long seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(seed, idx, 0, &state[idx]);
}

// CUDA kernel for Randomize matrix using cuRAND
__global__ void matrixRandomize(double* A, int rows, int cols, hiprandState* states) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < rows * cols) {
        // Generate random number using cuRAND
        A[idx] = hiprand_uniform(&states[idx]);
    }
}

// CUDA kernel for matrix addition
__global__ void matrixAdd(const double* __restrict__ A, const double* __restrict__ B, double* __restrict__ C, int rows, int cols) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < rows * cols) {
        C[idx] = A[idx] + B[idx];
    }
}

// CUDA kernel for matrix subtraction
__global__ void matrixSub(const double* __restrict__ A, const double* __restrict__ B, double* __restrict__ C, int rows, int cols) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < rows * cols) {
        C[idx] = A[idx] - B[idx];
    }
}

// CUDA kernel for matrix multiplication
__global__ void matrixMul(const double* __restrict__ A, const double* __restrict__ B, double* __restrict__ C, int rowsA, int colsA, int colsB) {
    // Tile size
    const int TILE_SIZE = 16;

    // Shared memory for tiles of A and B
    __shared__ double sharedA[TILE_SIZE][TILE_SIZE];
    __shared__ double sharedB[TILE_SIZE][TILE_SIZE];

    // Thread indices
    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;

    double sum = 0.0;

    // Loop over tiles
    for (int t = 0; t < (colsA + TILE_SIZE - 1) / TILE_SIZE; t++) {
        // Load tiles into shared memory
        if (row < rowsA && t * TILE_SIZE + threadIdx.x < colsA) {
            sharedA[threadIdx.y][threadIdx.x] = A[row * colsA + t * TILE_SIZE + threadIdx.x];
        } else {
            sharedA[threadIdx.y][threadIdx.x] = 0.0;
        }

        if (col < colsB && t * TILE_SIZE + threadIdx.y < colsA) {
            sharedB[threadIdx.y][threadIdx.x] = B[(t * TILE_SIZE + threadIdx.y) * colsB + col];
        } else {
            sharedB[threadIdx.y][threadIdx.x] = 0.0;
        }

        __syncthreads(); // Synchronize to ensure all threads have loaded their tiles

        // Compute partial sum for the tile
        for (int k = 0; k < TILE_SIZE; k++) {
            sum += sharedA[threadIdx.y][k] * sharedB[k][threadIdx.x];
        }

        __syncthreads(); // Synchronize before loading the next tile
    }

    // Write the result to global memory
    if (row < rowsA && col < colsB) {
        C[row * colsB + col] = sum;
    }
}

// CUDA kernel for matrix Hadamard product
__global__ void matrixHadamard(const double* __restrict__ A, const double* __restrict__ B, double* __restrict__ C, int rows, int cols) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < rows * cols) {
        C[idx] = A[idx] * B[idx];
    }
}

// Wrapper function for launching matrixRandomize kernel
extern "C" void launchMatrixRandomize(double* d_A, int rows, int cols) {
    int threadsPerBlock = 256; // Optimal for most GPUs
    int numElements = rows * cols;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;

    // Allocate memory for cuRAND states
    hiprandState* d_states;
    hipMalloc((void**)&d_states, numElements * sizeof(hiprandState));

    // Initialize cuRAND states
    setup_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_states, time(0));

    #ifdef DEBUG
    printf("Launching matrixRandomize kernel: blocksPerGrid = %d, threadsPerBlock = %d, numElements = %d\n",
           blocksPerGrid, threadsPerBlock, numElements);
    #endif

    // Launch the kernel
    matrixRandomize<<<blocksPerGrid, threadsPerBlock>>>(d_A, rows, cols, d_states);

    // Error checking
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(err));
        return;
    }

    // Synchronize to ensure the kernel completes
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        fprintf(stderr, "Kernel execution failed: %s\n", hipGetErrorString(err));
        return;
    }

    // Free cuRAND states
    hipFree(d_states);
}

// Wrapper function for launching matrixAdd kernel
extern "C" void launchMatrixAdd(double* d_A, double* d_B, double* d_C, int rows, int cols) {
    int threadsPerBlock = 256; // Optimal for most GPUs
    int blocksPerGrid = (rows * cols + threadsPerBlock - 1) / threadsPerBlock;

    #ifdef DEBUG
    printf("Launching matrixAdd kernel: blocksPerGrid = %d, threadsPerBlock = %d\n", blocksPerGrid, threadsPerBlock);
    #endif

    // Launch the kernel
    matrixAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, rows, cols);

    // Error checking
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(err));
        return;
    }

    // Synchronize to ensure the kernel completes
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        fprintf(stderr, "Kernel execution failed: %s\n", hipGetErrorString(err));
        return;
    }
}

// Wrapper function for launching matrixSub kernel
extern "C" void launchMatrixSub(double* d_A, double* d_B, double* d_C, int rows, int cols) {
    int threadsPerBlock = 256; // Optimal for most GPUs
    int blocksPerGrid = (rows * cols + threadsPerBlock - 1) / threadsPerBlock;

    #ifdef DEBUG
    printf("Launching matrixSub kernel: blocksPerGrid = %d, threadsPerBlock = %d\n", blocksPerGrid, threadsPerBlock);
    #endif

    // Launch the kernel
    matrixSub<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, rows, cols);

    // Check for kernel launch errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(err));
        return;
    }

    // Synchronize to ensure the kernel completes
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        fprintf(stderr, "Kernel execution failed: %s\n", hipGetErrorString(err));
        return;
    }
}

// Wrapper function for launching matrixMul kernel
extern "C" void launchMatrixMul(double* d_A, double* d_B, double* d_C, int rowsA, int colsA, int colsB) {
    dim3 threadsPerBlock(16, 16); // Optimal for shared memory tiles
    dim3 blocksPerGrid((colsB + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (rowsA + threadsPerBlock.y - 1) / threadsPerBlock.y);

    #ifdef DEBUG
    printf("Launching matrixMul kernel: blocksPerGrid = (%d, %d), threadsPerBlock = (%d, %d)\n",
           blocksPerGrid.x, blocksPerGrid.y, threadsPerBlock.x, threadsPerBlock.y);
    #endif
    
    // Launch the kernel
    matrixMul<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, rowsA, colsA, colsB);

    // Check for kernel launch errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(err));
        return;
    }

    // Synchronize to ensure the kernel completes
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        fprintf(stderr, "Kernel execution failed: %s\n", hipGetErrorString(err));
        return;
    }
}

// Wrapper function for launching matrixHadamard kernel
extern "C" void launchMatrixHadamard(double* d_A, double* d_B, double* d_C, int rows, int cols) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (rows * cols + threadsPerBlock - 1) / threadsPerBlock;

    #ifdef DEBUG
    printf("Launching matrixHadamard kernel: blocksPerGrid = %d, threadsPerBlock = %d\n", blocksPerGrid, threadsPerBlock);
    #endif

    // Launch the kernel
    matrixHadamard<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, rows, cols);

    // Check for kernel launch errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(err));
        return;
    }

    // Synchronize to ensure the kernel completes
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        fprintf(stderr, "Kernel execution failed: %s\n", hipGetErrorString(err));
        return;
    }
}