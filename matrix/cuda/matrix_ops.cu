#include <hip/hip_runtime.h>
#include <stdio.h>

// CUDA kernel for matrix addition
__global__ void matrixAdd(double* A, double* B, double* C, int rows, int cols) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < rows * cols) {
        C[idx] = A[idx] + B[idx];
    }
}

// CUDA kernel for matrix multiplication
__global__ void matrixMul(double* A, double* B, double* C, int rowsA, int colsA, int colsB) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rowsA && col < colsB) {
        double sum = 0.0;
        for (int k = 0; k < colsA; k++) {
            sum += A[row * colsA + k] * B[k * colsB + col];
        }
        C[row * colsB + col] = sum;
    }
}

// Wrapper function for launching matrixAdd kernel
extern "C" void launchMatrixAdd(double* d_A, double* d_B, double* d_C, int rows, int cols) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (rows * cols + threadsPerBlock - 1) / threadsPerBlock;

    printf("Launching matrixAdd kernel: blocksPerGrid = %d, threadsPerBlock = %d\n", blocksPerGrid, threadsPerBlock);

    // Launch the kernel
    matrixAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, rows, cols);

    // Check for kernel launch errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(err));
        return;
    }

    // Synchronize to ensure the kernel completes
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        fprintf(stderr, "Kernel execution failed: %s\n", hipGetErrorString(err));
        return;
    }
}

// Wrapper function for launching matrixMul kernel
extern "C" void launchMatrixMul(double* d_A, double* d_B, double* d_C, int rowsA, int colsA, int colsB) {
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((colsB + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (rowsA + threadsPerBlock.y - 1) / threadsPerBlock.y);

    printf("Launching matrixMul kernel: blocksPerGrid = (%d, %d), threadsPerBlock = (%d, %d)\n",
           blocksPerGrid.x, blocksPerGrid.y, threadsPerBlock.x, threadsPerBlock.y);

    // Launch the kernel
    matrixMul<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, rowsA, colsA, colsB);

    // Check for kernel launch errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(err));
        return;
    }

    // Synchronize to ensure the kernel completes
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        fprintf(stderr, "Kernel execution failed: %s\n", hipGetErrorString(err));
        return;
    }
}