#include <hip/hip_runtime.h>
#include <stdio.h>

// CUDA kernel for matrix addition
__global__ void matrixAdd(float* A, float* B, float* C, int rows, int cols) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < rows * cols) {
        C[idx] = A[idx] + B[idx];
    }
}

// CUDA kernel for matrix multiplication
__global__ void matrixMul(float* A, float* B, float* C, int rowsA, int colsA, int colsB) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rowsA && col < colsB) {
        float sum = 0.0f;
        for (int k = 0; k < colsA; k++) {
            sum += A[row * colsA + k] * B[k * colsB + col];
        }
        C[row * colsB + col] = sum;
    }
}

// CUDA kernel for element-wise multiplication (Hadamard product)
__global__ void matrixHadamard(float* A, float* B, float* C, int rows, int cols) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < rows * cols) {
        C[idx] = A[idx] * B[idx];
    }
}

// CUDA kernel for scalar multiplication
__global__ void matrixScalarMul(float* A, float scalar, float* C, int rows, int cols) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < rows * cols) {
        C[idx] = A[idx] * scalar;
    }
}